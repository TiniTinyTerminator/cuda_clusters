#include "cuda_kernel.h"

#include <hipfft/hipfft.h>
 

// Apply FFT to the path using cuFFT to determine Fourier coefficients
void apply_fft(const std::vector<Point>& path)
{
    size_t N = path.size();

    // Allocate device memory for the input and output data
    hipfftComplex* d_in;
    hipfftComplex* d_out;
    hipMalloc(&d_in, sizeof(hipfftComplex) * N);
    hipMalloc(&d_out, sizeof(hipfftComplex) * N);

    // Copy the path data to the device
    std::vector<hipfftComplex> h_in(N);
    for (size_t i = 0; i < N; ++i)
    {
        h_in[i].x = static_cast<float>(path[i].x); // Real part
        h_in[i].y = static_cast<float>(path[i].y); // Imaginary part
    }
    hipMemcpy(d_in, h_in.data(), sizeof(hipfftComplex) * N, hipMemcpyHostToDevice);

    // Create a cuFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

    // Execute the FFT
    hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD);

    // Copy the results back to the host
    std::vector<hipfftComplex> h_out(N);
    hipMemcpy(h_out.data(), d_out, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost);

    // Output the Fourier coefficients
    std::cout << "Fourier Coefficients: " << std::endl;
    for (size_t i = 0; i < N; ++i)
    {
        std::cout << "Coefficient " << i << ": (" << h_out[i].x << ", " << h_out[i].y << ")" << std::endl;
    }

    // Destroy the cuFFT plan and free device memory
    hipfftDestroy(plan);
    hipFree(d_in);
    hipFree(d_out);
}